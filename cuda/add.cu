
#include <hip/hip_runtime.h>
#include <stdio.h>

const int blockSize = 900;

__global__ void add(int* a, int* b, int* c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

__global__ void vectorAdd(int* a, int* b, int n, int* out) {
  int sum = n * a[threadIdx.x] * b[threadIdx.x];
  atomicAdd(out, sum);
}

int main(void) {
  int a[blockSize];
  int b[blockSize];
  int c;
  int *d_a, *d_b, *d_c;
  int size = blockSize * sizeof(int);

  for (int i = 0; i < blockSize; ++i) {
    a[i] = 1;
    b[i] = 1;
  }

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, sizeof(int));

  hipMemset(d_c, 0, sizeof(int));

  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  add<<<1, blockSize>>>(d_a, d_b, d_b);
  vectorAdd<<<1, blockSize>>>(d_a, d_b, 5, d_c);

  hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

  printf("%d\n", c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
